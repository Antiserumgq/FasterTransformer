/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <float.h>

#include "src/fastertransformer/kernels/sampling_topk_kernels.h"
#include "src/fastertransformer/kernels/sampling_topp_kernels.h"
#include "src/fastertransformer/layers/sampling_layers/TopKSamplingLayer.h"

namespace fastertransformer {

template<typename T>
void TopKSamplingLayer<T>::allocateBuffer()
{
    if (is_allocate_buffer_ == false) {
        invokeTopKSampling<T>(nullptr,
                              sampling_workspace_size_,
                              nullptr,
                              nullptr,
                              nullptr,
                              nullptr,
                              nullptr,
                              nullptr,
                              top_k_,
                              vocab_size_padded_,
                              end_id_,
                              stream_,
                              max_batch_size_);
        sampling_workspace_ = allocator_->malloc(sampling_workspace_size_, false);
        curandstate_buf_ =
            reinterpret_cast<hiprandState_t*>(allocator_->malloc(sizeof(hiprandState_t) * max_batch_size_, false));
        invokeInitialize();
        is_allocate_buffer_ = true;
    }
}

template<typename T>
void TopKSamplingLayer<T>::freeBuffer()
{
    if (is_allocate_buffer_ == true) {
        allocator_->free(sampling_workspace_);
        allocator_->free(curandstate_buf_);
        is_allocate_buffer_ = false;
    }
}

template<typename T>
void TopKSamplingLayer<T>::invokeInitialize()
{
    invokeCurandInitialize(curandstate_buf_, max_batch_size_, random_seed_, stream_);
    sync_check_cuda_error();
}

template<typename T>
void TopKSamplingLayer<T>::runSampling(std::vector<fastertransformer::Tensor>* output_tensors,
                                       const std::vector<fastertransformer::Tensor>* input_tensors)
{
    // input_tensors:
    //      logits [local_batch_size, vocab_size_padded]
    //      embedding_bias [vocab_size_padded]
    //      step [1] on cpu
    //      max_input_length [1] on cpu
    //      input_lengths [local_batch_size]
    //      ite [1] on cpu

    // output_tensors:
    //      output_ids [max_seq_len, batch_size]
    //      finished [local_batch_size]
    //      sequence_length [local_batch_size]
    //      cum_log_probs [local_batch_size], must be float*

    FT_CHECK(input_tensors->size() == 6);
    FT_CHECK(output_tensors->size() == 4);

    const int batch_size = output_tensors->at(0).shape[1];
    const int local_batch_size = input_tensors->at(0).shape[0];
    const int step = *((int*)input_tensors->at(2).data);
    const int ite = *((int*)input_tensors->at(5).data);

    invokeAddBiasEndMask((T*)(input_tensors->at(0).data),
                         (T*)(nullptr),
                         end_id_,
                         (bool*)output_tensors->at(1).data,
                         local_batch_size,
                         vocab_size_padded_,
                         stream_);
    sync_check_cuda_error();

    if (output_tensors->at(3).data != nullptr) {
        invokeAddBiasSoftMax((T*)(input_tensors->at(0).data),
                             (T*)(nullptr),
                             end_id_,
                             (bool*)output_tensors->at(1).data,
                             local_batch_size,
                             vocab_size_padded_,
                             vocab_size_,
                             stream_);
    }

    invokeTopKSampling(sampling_workspace_,
                       sampling_workspace_size_,
                       (T*)(input_tensors->at(0).data),
                       ((int*)output_tensors->at(0).data) + step * batch_size + ite * local_batch_size,
                       (int*)output_tensors->at(2).data,
                       (bool*)output_tensors->at(1).data,
                       (float*)output_tensors->at(3).data,
                       curandstate_buf_ + ite * local_batch_size,
                       top_k_,
                       vocab_size_padded_,
                       end_id_,
                       stream_,
                       local_batch_size);
    sync_check_cuda_error();
}

template<typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(size_t max_batch_size,
                                        size_t vocab_size,
                                        size_t vocab_size_padded,
                                        int end_id,
                                        size_t top_k,
                                        unsigned long long random_seed,
                                        float temperature,
                                        float len_penalty,
                                        float repetition_penalty,
                                        hipStream_t stream,
                                        cublasMMWrapper* cublas_wrapper,
                                        IAllocator* allocator,
                                        bool is_free_buffer_after_forward):
    BaseSamplingLayer<T>(max_batch_size,
                         vocab_size,
                         vocab_size_padded,
                         end_id,
                         top_k,
                         random_seed,
                         0.0f,
                         temperature,
                         len_penalty,
                         repetition_penalty,
                         stream,
                         cublas_wrapper,
                         allocator,
                         is_free_buffer_after_forward,
                         nullptr)
{
}

template<typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(TopKSamplingLayer<T> const& top_k_sampling_layer):
    BaseSamplingLayer<T>(top_k_sampling_layer)
{
}

template<typename T>
TopKSamplingLayer<T>::~TopKSamplingLayer()
{
    freeBuffer();
}

template class TopKSamplingLayer<float>;
template class TopKSamplingLayer<half>;

}  // namespace fastertransformer