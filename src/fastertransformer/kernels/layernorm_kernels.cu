#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/kernels/layernorm_kernels.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"

namespace fastertransformer {

// TODO(bhsueh) add half2 implementation
template<typename T, int N>
__global__ void
addBiasResidualPostLayerNorm(T* out, const T* input, const T* bias, const T* gamma, const T* beta, int m, int n)
{
    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;
    float local_out_cache[N];

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out = (float)(out[blockIdx.x * n + idx] + input[blockIdx.x * n + idx] + __ldg(&bias[idx]));
        mean += local_out;
        // save local_out to local_out_cache to save some recompute
        local_out_cache[i] = local_out;
        idx += blockDim.x;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out = local_out_cache[i];
        variance += (local_out - s_mean) * (local_out - s_mean);
        idx += blockDim.x;
    }
    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0)
        s_variance = variance / n + 1e-6f;
    __syncthreads();

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out = local_out_cache[i];
        out[blockIdx.x * n + idx] =
            (T)(((local_out - s_mean) * rsqrtf(s_variance)) * (float)(__ldg(&gamma[idx])) + (float)(__ldg(&beta[idx])));
        idx += blockDim.x;
    }
}

template<int N>
__global__ void addBiasResidualPostLayerNormHalf(
    half* out, const half* input, const half* bias, const half* gamma, const half* beta, int m, int n)
{
    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    half2* out_ptr = (half2*)out;
    const half2* input_ptr = (const half2*)input;
    const half2* bias_ptr = (const half2*)bias;
    const half2* gamma_ptr = (const half2*)gamma;
    const half2* beta_ptr = (const half2*)beta;

    float2 out_fp2_cache[N];

    float local_out = 0.0f;
#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n / 2 && i < N; ++i) {
        int id = blockIdx.x * n / 2 + idx;
        float2 local_out_fp2 = __half22float2(__hadd2(__hadd2(out_ptr[id], input_ptr[id]), __ldg(&bias_ptr[idx])));
        local_out += local_out_fp2.x;
        local_out += local_out_fp2.y;
        // save local_out_fp2 to out_fp2_cache to save some recomputation
        out_fp2_cache[i] = local_out_fp2;
        idx += blockDim.x;
    }

    mean = blockReduceSum<float>(local_out);
    if (threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; i < N && idx < n / 2; ++i) {
        float2 local_out_fp2 = out_fp2_cache[i];
        variance += (local_out_fp2.x - s_mean) * (local_out_fp2.x - s_mean);
        variance += (local_out_fp2.y - s_mean) * (local_out_fp2.y - s_mean);
        idx += blockDim.x;
    }

    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; i < N && idx < n / 2; ++i) {
        int id = blockIdx.x * n / 2 + idx;
        float2 local_out_fp2 = out_fp2_cache[i];
        float2 gamma_val = __half22float2(__ldg(&gamma_ptr[idx]));
        float2 beta_val = __half22float2(__ldg(&beta_ptr[idx]));
        local_out_fp2.x = (local_out_fp2.x - s_mean) * s_variance * gamma_val.x + beta_val.x;
        local_out_fp2.y = (local_out_fp2.y - s_mean) * s_variance * gamma_val.y + beta_val.y;
        out_ptr[id] = __float22half2_rn(local_out_fp2);
        idx += blockDim.x;
    }
}

template<typename T>
__global__ void
generalAddBiasResidualPostLayerNorm(T* out, const T* input, const T* bias, const T* gamma, const T* beta, int m, int n)
{
    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out = (float)(out[blockIdx.x * n + idx] + input[blockIdx.x * n + idx] + __ldg(&bias[idx]));
        mean += local_out;
        // save local_out to out to save some recompute
        out[blockIdx.x * n + idx] = local_out;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out = out[blockIdx.x * n + idx];
        variance += (local_out - s_mean) * (local_out - s_mean);
    }
    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0)
        s_variance = variance / n + 1e-6f;
    __syncthreads();

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out = out[blockIdx.x * n + idx];
        out[blockIdx.x * n + idx] =
            (T)(((local_out - s_mean) * rsqrtf(s_variance)) * (float)(__ldg(&gamma[idx])) + (float)(__ldg(&beta[idx])));
    }
}

template<>
__global__ void generalAddBiasResidualPostLayerNorm(
    half* out, const half* input, const half* bias, const half* gamma, const half* beta, int m, int n)
{
    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    half2* out_ptr = (half2*)out;
    const half2* input_ptr = (const half2*)input;
    const half2* bias_ptr = (const half2*)bias;
    const half2* gamma_ptr = (const half2*)gamma;
    const half2* beta_ptr = (const half2*)beta;

    float local_out = 0.0f;
    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int id = blockIdx.x * n / 2 + idx;
        half2 tmp = __hadd2(__hadd2(out_ptr[id], input_ptr[id]), __ldg(&bias_ptr[idx]));
        float2 local_out_fp2 = __half22float2(tmp);
        local_out += local_out_fp2.x;
        local_out += local_out_fp2.y;
        // save tmp to out_ptr to save some recomputation
        out_ptr[id] = tmp;
    }

    mean = blockReduceSum<float>(local_out);
    if (threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int id = blockIdx.x * n / 2 + idx;
        float2 local_out_fp2 = __half22float2(out_ptr[id]);
        variance += (local_out_fp2.x - s_mean) * (local_out_fp2.x - s_mean);
        variance += (local_out_fp2.y - s_mean) * (local_out_fp2.y - s_mean);
    }

    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int id = blockIdx.x * n / 2 + idx;
        float2 local_out_fp2 = __half22float2(out_ptr[id]);
        float2 gamma_val = __half22float2(__ldg(&gamma_ptr[idx]));
        float2 beta_val = __half22float2(__ldg(&beta_ptr[idx]));
        local_out_fp2.x = (local_out_fp2.x - s_mean) * s_variance * gamma_val.x + beta_val.x;
        local_out_fp2.y = (local_out_fp2.y - s_mean) * s_variance * gamma_val.y + beta_val.y;
        out_ptr[id] = __float22half2_rn(local_out_fp2);
    }
}

template<typename T>
__global__ void addBiasResidualPostLayerNormV2(T* out,
                                               const T* __restrict input,
                                               const T* __restrict bias,
                                               const T* __restrict gamma,
                                               const T* __restrict beta,
                                               int n)
{
    const int ite = 4;
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;
    float local_out[ite];

    float sum = 0.0f;
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id = i * blockDim.x + tid;
        int id = bid * n + col_id;
        local_out[i] = (float)(out[id] + __ldg(&input[id]) + __ldg(&bias[col_id]));
        sum += local_out[i];
    }

    mean = blockReduceSum<float>(sum);
    if (tid == 0)
        s_mean = mean / n;
    __syncthreads();

    float var = 0.0f;
#pragma unroll
    for (int i = 0; i < ite; i++) {
        float diff = local_out[i] - s_mean;
        var += diff * diff;
    }

    variance = blockReduceSum<float>(var);
    if (tid == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id = i * blockDim.x + tid;
        int id = bid * n + col_id;
        out[id] =
            (T)((local_out[i] - s_mean) * s_variance * (float)__ldg(&gamma[col_id]) + (float)__ldg(&beta[col_id]));
    }
}

template<>
__global__ void addBiasResidualPostLayerNormV2(half* out,
                                               const half* __restrict input,
                                               const half* __restrict bias,
                                               const half* __restrict gamma,
                                               const half* __restrict beta,
                                               int n)
{
    const int ite = 4;
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;
    half2 local_out_half2[ite];

    half2* out_ptr = (half2*)out;
    const half2* input_ptr = (const half2*)input;
    const half2* bias_ptr = (const half2*)bias;
    const half2* gamma_ptr = (const half2*)gamma;
    const half2* beta_ptr = (const half2*)beta;

    // float sum = 0.0f;
    half2 sum = __float2half2_rn(0.0f);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id = i * blockDim.x + tid;
        int id = bid * n / 2 + col_id;
        local_out_half2[i] = out_ptr[id] + __ldg(&input_ptr[id]) + __ldg(&bias_ptr[col_id]);
        sum += local_out_half2[i];
    }

    mean = blockReduceSum<float>((float)(sum.x + sum.y));
    if (threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    float var = 0.0f;
    half2 s_mean_2 = __float2half2_rn(s_mean);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        local_out_half2[i] = local_out_half2[i] - s_mean_2;
        float v1 = (float)local_out_half2[i].x;
        float v2 = (float)local_out_half2[i].y;
        var += v1 * v1 + v2 * v2;
    }

    variance = blockReduceSum<float>(var);
    if (threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    half2 s_var_2 = __float2half2_rn(s_variance);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id = i * blockDim.x + tid;
        int id = bid * n / 2 + col_id;
        out_ptr[id] = local_out_half2[i] * s_var_2 * __ldg(&gamma_ptr[col_id]) + __ldg(&beta_ptr[col_id]);
    }
}

template<typename T>
void invokeAddBiasResidualLayerNorm(
    T* out, const T* input, const T* bias, const T* gamma, const T* beta, int m, int n, hipStream_t stream)
{
    dim3 grid(m);
    dim3 block(std::min(n, 1024));
    if (n == 768 || n == 1024) {
        addBiasResidualPostLayerNormV2<T><<<grid, n / 4, 0, stream>>>(out, input, bias, gamma, beta, n);
    }
    else {
        block.x = std::min(n, 1024);
        int num_trips = (n + block.x - 1) / block.x;
        if (num_trips == 1) {
            addBiasResidualPostLayerNorm<T, 1><<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
        }
        else if (num_trips == 2) {
            addBiasResidualPostLayerNorm<T, 2><<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
        }
        else {
            generalAddBiasResidualPostLayerNorm<T><<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
        }
    }
}

template<>
void invokeAddBiasResidualLayerNorm(half* out,
                                    const half* input,
                                    const half* bias,
                                    const half* gamma,
                                    const half* beta,
                                    int m,
                                    int n,
                                    hipStream_t stream)
{
    dim3 grid(m);
    dim3 block(std::min(n, 1024));

    if (m >= 512 && (n == 768 || n == 1024)) {
        addBiasResidualPostLayerNormV2<half><<<grid, n / 8, 0, stream>>>(out, input, bias, gamma, beta, n);
    }
    else {
        block.x = std::min(n, 1024);
        int num_trips = (n + block.x - 1) / block.x;
        if (num_trips == 1) {
            addBiasResidualPostLayerNorm<half, 1><<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
        }
        else if (num_trips == 2) {
            addBiasResidualPostLayerNorm<half, 2><<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
        }
        else {
            generalAddBiasResidualPostLayerNorm<half><<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
        }
    }
}

template void invokeAddBiasResidualLayerNorm(float* out,
                                             const float* input,
                                             const float* bias,
                                             const float* gamma,
                                             const float* beta,
                                             int m,
                                             int n,
                                             hipStream_t stream);
template void invokeAddBiasResidualLayerNorm(half* out,
                                             const half* input,
                                             const half* bias,
                                             const half* gamma,
                                             const half* beta,
                                             int m,
                                             int n,
                                             hipStream_t stream);

template<typename T>
__global__ void generalAddBiasResidualLayerNorm(const T* __restrict input,
                                                const T* __restrict gamma,
                                                const T* __restrict beta,
                                                const T* __restrict bias,
                                                T* output,
                                                T* norm_output,
                                                int m,
                                                int n)
{
    int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    float local_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float local_out = (float)(__ldg(&input[blockIdx.x * n + i]));
        local_out += (float)(output[blockIdx.x * n + i]);
        if (bias != nullptr) {
            local_out += (float)(__ldg(&bias[i]));
        }
        output[blockIdx.x * n + i] = (T)local_out;
        local_sum += local_out;
    }

    mean = blockReduceSum(local_sum);

    if (threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    float local_var_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float diff = (float)(output[blockIdx.x * n + i]) - s_mean;
        local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6);
    __syncthreads();

    for (int i = tid; i < n; i += blockDim.x) {
        float beta_val = (beta == nullptr) ? 0.0f : (float)(__ldg(&beta[i]));
        norm_output[blockIdx.x * n + i] =
            (T)((((float)output[blockIdx.x * n + i] - s_mean) * s_variance) * (float)(__ldg(&gamma[i])) + beta_val);
    }
}

template<typename T>
void invokeGeneralAddBiasResidualPreLayerNorm(T* output,
                                              T* norm_output,
                                              const T* input,
                                              const T* gamma,
                                              const T* beta,
                                              const T* bias,
                                              int m,
                                              int n,
                                              hipStream_t stream)
{
    dim3 grid(m);
    dim3 block(min(n, 1024));

    /* For general cases, n is equal to hidden_units, e.g., 512/1024.
    Since we have warp shuffle inside the code, block.x % 32 should be 0.
    */

    if (n % 32 != 0)
        block.x = 1024;

    block.x = block.x / (4 / sizeof(T));  // if using half, only need half of block.x

    /* should pay attention to the rsqrt precision*/
    generalAddBiasResidualLayerNorm<T>
        <<<grid, block, 0, stream>>>(input, gamma, beta, bias, output, norm_output, m, n);  // For gpt-3
}

template void invokeGeneralAddBiasResidualPreLayerNorm(float* output,
                                                       float* norm_output,
                                                       const float* input,
                                                       const float* gamma,
                                                       const float* beta,
                                                       const float* bias,
                                                       int m,
                                                       int n,
                                                       hipStream_t stream);

template void invokeGeneralAddBiasResidualPreLayerNorm(half* output,
                                                       half* norm_output,
                                                       const half* input,
                                                       const half* gamma,
                                                       const half* beta,
                                                       const half* bias,
                                                       int m,
                                                       int n,
                                                       hipStream_t stream);

template<typename T>
__global__ void generalAddResidualT5LayerNorm(
    const T* __restrict input, const T* __restrict gamma, T* output, T* norm_output, int m, int n)
{
    // layernorm module in the T5 style No bias and no subtraction of mean.
    __shared__ float s_variance;
    float variance = 0.0f;

    float local_var_sum = 0.0f;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        output[blockIdx.x * n + i] =
            clamp_inf_for_half<T>((float)__ldg(&input[blockIdx.x * n + i]) + (float)output[blockIdx.x * n + i]);

        float diff = (float)(output[blockIdx.x * n + i]);
        local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n + 1e-6);
    }
    __syncthreads();

    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float out_val = (((float)output[blockIdx.x * n + i]) * s_variance) * (float)(__ldg(&gamma[i]));
        norm_output[blockIdx.x * n + i] =
            clamp_inf_for_half<T>((((float)output[blockIdx.x * n + i]) * s_variance) * (float)(__ldg(&gamma[i])));
    }
}

template<typename T>
void invokeGeneralAddResidualT5PreLayerNorm(
    T* output, T* norm_output, const T* input, const T* gamma, int m, int n, hipStream_t stream)
{
    dim3 grid(m);
    dim3 block(min(n, 1024));

    /* For general cases, n is equal to hidden_units, e.g., 512/1024.
    Since we have warp shuffle inside the code, block.x % 32 should be 0.
    */

    if (n % 32 != 0) {
        block.x = 1024;
    }

    block.x = block.x / (4 / sizeof(T));  // if using half, only need half of block.x

    /* should pay attention to the rsqrt precision*/
    generalAddResidualT5LayerNorm<T><<<grid, block, 0, stream>>>(input, gamma, output, norm_output, m, n);
}

template void invokeGeneralAddResidualT5PreLayerNorm(
    float* output, float* norm_output, const float* input, const float* gamma, int m, int n, hipStream_t stream);

template void invokeGeneralAddResidualT5PreLayerNorm(
    half* output, half* norm_output, const half* input, const half* gamma, int m, int n, hipStream_t stream);

template<typename T>
__global__ void generalLayerNorm(
    const T* __restrict input, const T* __restrict gamma, const T* __restrict beta, T* output, int m, int n)
{
    const int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    float local_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        local_sum += (float)(__ldg(&input[blockIdx.x * n + i]));
    }

    mean = blockReduceSum(local_sum);

    if (threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    float local_var_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float diff = (float)(__ldg(&input[blockIdx.x * n + i])) - s_mean;
        local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6);
    __syncthreads();

    for (int i = tid; i < n; i += blockDim.x) {
        float beta_val = (beta == nullptr) ? 0.0f : (float)__ldg(&beta[i]);
        output[blockIdx.x * n + i] =
            (T)((((float)input[blockIdx.x * n + i] - s_mean) * s_variance) * (float)(__ldg(&gamma[i])) + beta_val);
    }
}

template<typename T>
void invokeGeneralLayerNorm(
    T* out, const T* input, const T* gamma, const T* beta, const int m, const int n, hipStream_t stream)
{
    dim3 grid(m);
    dim3 block(min(n, 1024));

    /* For general cases, n is equal to hidden_units, e.g., 512/1024.
        Since we have warp shuffle inside the code, block.x % 32 should be 0.
    */
    if (n % 32 != 0)
        block.x = 1024;

    block.x = block.x / (4 / sizeof(T));  // if using half, only need half of block.x

    /* should pay attention to the rsqrt precision*/
    generalLayerNorm<T><<<grid, block, 0, stream>>>(input, gamma, beta, out, m, n);  // For gpt-3
}

template void invokeGeneralLayerNorm(float* out,
                                     const float* input,
                                     const float* gamma,
                                     const float* beta,
                                     const int m,
                                     const int n,
                                     hipStream_t stream);
template void invokeGeneralLayerNorm(
    half* out, const half* input, const half* gamma, const half* beta, const int m, const int n, hipStream_t stream);

template<typename T>
__global__ void generalT5LayerNorm(const T* __restrict input, const T* __restrict gamma, T* output, int m, int n)
{
    // layernorm module in the T5 style No bias and no subtraction of mean.
    const int tid = threadIdx.x;

    __shared__ float s_variance;
    float variance = 0.0f;

    float local_var_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float diff = (float)(__ldg(&input[blockIdx.x * n + i]));
        local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6);
    __syncthreads();

    for (int i = tid; i < n; i += blockDim.x) {
        output[blockIdx.x * n + i] =
            clamp_inf_for_half<T>((((float)input[blockIdx.x * n + i]) * s_variance) * (float)(__ldg(&gamma[i])));
    }
}

template<typename T>
void invokeGeneralT5LayerNorm(T* out, const T* input, const T* gamma, const int m, const int n, hipStream_t stream)
{
    dim3 grid(m);
    dim3 block(min(n, 1024));

    /* For general cases, n is equal to hidden_units, e.g., 512/1024.
        Since we have warp shuffle inside the code, block.x % 32 should be 0.
    */
    if (n % 32 != 0)
        block.x = 1024;

    block.x = block.x / (4 / sizeof(T));  // if using half, only need half of block.x

    /* should pay attention to the rsqrt precision*/
    generalT5LayerNorm<T><<<grid, block, 0, stream>>>(input, gamma, out, m, n);  // For gpt-3
}

template void invokeGeneralT5LayerNorm(
    float* out, const float* input, const float* gamma, const int m, const int n, hipStream_t stream);
template void invokeGeneralT5LayerNorm(
    half* out, const half* input, const half* gamma, const int m, const int n, hipStream_t stream);

}  // namespace fastertransformer