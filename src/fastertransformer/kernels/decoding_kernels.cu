#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/kernels/decoding_kernels.h"

namespace fastertransformer {

static const float HALF_FLT_MAX = 65504.F;

template<typename T>
__global__ void decodingInitialize(bool* finished,
                                   int* sequence_length,
                                   int* word_ids,
                                   T* cum_log_probs,
                                   const int sentence_id,
                                   const int batch_size,
                                   const int beam_width,
                                   const int max_input_length)
{
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : 1e20f;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * beam_width;
         index += blockDim.x * gridDim.x) {
        finished[index] = false;
        sequence_length[index] = max_input_length;
        if (word_ids != nullptr) {
            word_ids[index] = sentence_id;
        }
        cum_log_probs[index] = (index % beam_width == 0) ? (T)0.0f : -MAX_T_VAL;
    }
}

template<typename T>
void invokeDecodingInitialize(bool* finished,
                              int* sequence_length,
                              int* word_ids,
                              T* cum_log_probs,
                              const int sentence_id,
                              const int batch_size,
                              const int beam_width,
                              const int max_input_length,
                              hipStream_t stream)
{
    dim3 grid((int)ceil(batch_size * beam_width * 1.0 / 256));
    dim3 block(256);

    decodingInitialize<T><<<grid, block, 0, stream>>>(
        finished, sequence_length, word_ids, cum_log_probs, sentence_id, batch_size, beam_width, max_input_length);
}

template void invokeDecodingInitialize(bool* finished,
                                       int* sequence_length,
                                       int* word_ids,
                                       float* cum_log_probs,
                                       const int sentence_id,
                                       const int batch_size,
                                       const int beam_width,
                                       const int max_input_length,
                                       hipStream_t stream);

template void invokeDecodingInitialize(bool* finished,
                                       int* sequence_length,
                                       int* word_ids,
                                       half* cum_log_probs,
                                       const int sentence_id,
                                       const int batch_size,
                                       const int beam_width,
                                       const int max_input_length,
                                       hipStream_t stream);

template<typename T>
__global__ void embeddingLookupPosEncoding(T* from_tensor,
                                           const T* embedding_table,
                                           const T* position_encoding,
                                           const int* all_ids,
                                           const int* input_lengths,
                                           const int local_batch_size,
                                           const int hidden_units,
                                           const int step,
                                           const int max_input_length,
                                           const int batch_size,
                                           const int ite,
                                           const T scale)
{
    // 1. lookup from embedding table
    // 2. multiply scale
    // 3. add the position encoding
    const int id_offset = step * batch_size + ite * local_batch_size;

    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < local_batch_size * hidden_units;
         index += blockDim.x * gridDim.x) {
        const int row_index = index / hidden_units;
        const int col_index = index % hidden_units;
        const int step_offset = input_lengths == nullptr ?
                                    step * hidden_units :
                                    (step - max_input_length + input_lengths[row_index]) * hidden_units;
        T val = embedding_table[all_ids[id_offset + row_index] * hidden_units + col_index] * scale;
        if (position_encoding != nullptr) {
            val = val + position_encoding[step_offset + col_index];
        }
        from_tensor[index] = val;
    }
}

template<typename T>
void invokeEmbeddingLookupPosEncoding(T* from_tensor,
                                      const T* embedding_table,
                                      const T* position_encoding,
                                      const int* all_ids,
                                      const int* input_lengths,
                                      const int local_batch_size,
                                      const int hidden_units,
                                      const T scale,
                                      const int step,
                                      const int max_input_length,
                                      const int batch_size,
                                      const int ite,
                                      hipStream_t stream)
{
    dim3 grid(min(local_batch_size, 65536));
    dim3 block(min(hidden_units, 1024));
    embeddingLookupPosEncoding<T><<<grid, block, 0, stream>>>(from_tensor,
                                                              embedding_table,
                                                              position_encoding,
                                                              all_ids,
                                                              input_lengths,
                                                              local_batch_size,
                                                              hidden_units,
                                                              step,
                                                              max_input_length,
                                                              batch_size,
                                                              ite,
                                                              scale);
}

template void invokeEmbeddingLookupPosEncoding(float* from_tensor,
                                               const float* embedding_table,
                                               const float* position_encoding,
                                               const int* all_ids,
                                               const int* input_lengths,
                                               const int local_batch_size,
                                               const int hidden_units,
                                               const float scale,
                                               const int step,
                                               const int max_input_length,
                                               const int batch_size,
                                               const int ite,
                                               hipStream_t stream);

template void invokeEmbeddingLookupPosEncoding(half* from_tensor,
                                               const half* embedding_table,
                                               const half* position_encoding,
                                               const int* all_ids,
                                               const int* input_lengths,
                                               const int local_batch_size,
                                               const int hidden_units,
                                               const half scale,
                                               const int step,
                                               const int max_input_length,
                                               const int batch_size,
                                               const int ite,
                                               hipStream_t stream);

template<typename T>
__global__ void paddingEmbedding(T* padded_embedding_kernel,
                                 T* padded_embedding_bias,
                                 const T* embedding_kernel,
                                 const T* embedding_bias,
                                 const int hidden_unit,
                                 const int vocab_size,
                                 const int vocab_size_padded)
{
    for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < hidden_unit * vocab_size_padded;
         id += blockDim.x * gridDim.x) {
        int row_id = id / vocab_size_padded;
        int col_id = id % vocab_size_padded;
        if (col_id < vocab_size) {
            padded_embedding_kernel[id] = embedding_kernel[row_id * vocab_size + col_id];
        }
        else {
            padded_embedding_kernel[id] = (T)(0.0f);
        }
    }

    for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < vocab_size_padded; id += blockDim.x * gridDim.x) {
        if (id < vocab_size) {
            padded_embedding_bias[id] = embedding_bias[id];
        }
        else {
            padded_embedding_bias[id] = (T)(0.0f);
        }
    }
}

template<typename T>
void invokePaddingEmbedding(T* padded_embedding_kernel,
                            T* padded_embedding_bias,
                            const T* embedding_kernel,
                            const T* embedding_bias,
                            const int hidden_unit,
                            const int vocab_size,
                            const int vocab_size_padded,
                            hipStream_t stream)
{
    dim3 block(512);
    dim3 grid((int)(ceil(hidden_unit * vocab_size_padded / 512.)));
    paddingEmbedding<<<grid, block, 0, stream>>>(padded_embedding_kernel,
                                                 padded_embedding_bias,
                                                 embedding_kernel,
                                                 embedding_bias,
                                                 hidden_unit,
                                                 vocab_size,
                                                 vocab_size_padded);
}

template void invokePaddingEmbedding(float* padded_embedding_kernel,
                                     float* padded_embedding_bias,
                                     const float* embedding_kernel,
                                     const float* embedding_bias,
                                     const int hidden_unit,
                                     const int vocab_size,
                                     const int vocab_size_padded,
                                     hipStream_t stream);

template void invokePaddingEmbedding(half* padded_embedding_kernel,
                                     half* padded_embedding_bias,
                                     const half* embedding_kernel,
                                     const half* embedding_bias,
                                     const int hidden_unit,
                                     const int vocab_size,
                                     const int vocab_size_padded,
                                     hipStream_t stream);

template<typename T>
__global__ void paddingEmbeddingKernel(T* padded_embedding_kernel,
                                       const T* embedding_kernel,
                                       const int hidden_unit,
                                       const int vocab_size,
                                       const int vocab_size_padded)
{
    for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < hidden_unit * vocab_size_padded;
         id += blockDim.x * gridDim.x) {
        int row_id = id / vocab_size_padded;
        int col_id = id % vocab_size_padded;
        if (col_id < vocab_size) {
            padded_embedding_kernel[id] = embedding_kernel[row_id * vocab_size + col_id];
        }
        else {
            padded_embedding_kernel[id] = (T)(0.0f);
        }
    }
}

template<typename T>
void invokePaddingEmbeddingKernel(T* padded_embedding_kernel,
                                  const T* embedding_kernel,
                                  const int hidden_unit,
                                  const int vocab_size,
                                  const int vocab_size_padded,
                                  hipStream_t stream)
{
    dim3 block(512);
    dim3 grid((int)(ceil(hidden_unit * vocab_size_padded / 512.)));
    paddingEmbeddingKernel<<<grid, block, 0, stream>>>(
        padded_embedding_kernel, embedding_kernel, hidden_unit, vocab_size, vocab_size_padded);
}

template void invokePaddingEmbeddingKernel(float* padded_embedding_kernel,
                                           const float* embedding_kernel,
                                           const int hidden_unit,
                                           const int vocab_size,
                                           const int vocab_size_padded,
                                           hipStream_t stream);

template void invokePaddingEmbeddingKernel(half* padded_embedding_kernel,
                                           const half* embedding_kernel,
                                           const int hidden_unit,
                                           const int vocab_size,
                                           const int vocab_size_padded,
                                           hipStream_t stream);

// modified from TensorFlow's implementation of tf.contrib.seq2seq.gather_tree
__global__ void gatherTree(int* beams,
                           int* max_sequence_lengths,
                           const int max_time,
                           const int batch_size,
                           const int beam_width,
                           const int* step_ids,
                           const int* parent_ids,
                           const int end_token)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size * beam_width; i += gridDim.x * blockDim.x) {
        const int batch = i / beam_width;
        const int beam = i % beam_width;

        // TODO(bhsueh) optimize the reduce_max operation for large beam_width
        int max_len = -1;
        for (int j = 0; j < beam_width; j++) {
            max_len = max(max_len, __ldg(max_sequence_lengths + batch * beam_width + j));
        }
        const int max_seq_len_b = min(max_time, max_len);
        if (max_seq_len_b <= 0) {
            continue;
        }

#define GET_IX(time_ix, beam_ix) (batch_size * beam_width * (time_ix) + beam_width * batch + (beam_ix))

        const int initial_beam_ix = GET_IX(max_seq_len_b - 1, beam);
        beams[initial_beam_ix] = __ldg(step_ids + initial_beam_ix);
        int parent = __ldg(parent_ids + initial_beam_ix) % beam_width;
        bool found_bad = false;

        for (int level = max_seq_len_b - 2; level >= 0; --level) {
            const int level_beam_ix = GET_IX(level, beam);
            const int level_parent_ix = GET_IX(level, parent);
            if (parent < 0 || parent > beam_width) {
                // beams[level_beam_ix] = -1;
                beams[level_beam_ix] = end_token;
                parent = -1;
                found_bad = true;
            }
            else {
                beams[level_beam_ix] = __ldg(step_ids + level_parent_ix);
                parent = __ldg(parent_ids + level_parent_ix) % beam_width;
            }
        }
        for (int level = max_seq_len_b; level < max_time; ++level) {
            const int level_beam_ix = GET_IX(level, beam);
            beams[level_beam_ix] = end_token;
        }

        // Not necessary when using a BeamSearchDecoder, but necessary
        // when a user feeds in possibly broken trajectory (i.e., non-eos
        // entries in a beam following eos entries).
        if (!found_bad) {
            bool finished = false;
            for (int time = 0; time < max_seq_len_b; ++time) {
                const int level_beam_ix = GET_IX(time, beam);
                if (finished) {
                    beams[level_beam_ix] = end_token;
                }
                else if (beams[level_beam_ix] == end_token) {
                    finished = true;
                }
            }
        }
#undef GET_IX
    }
}

void invokeGatherTree(int* beams,
                      int* max_sequence_lengths,
                      const int max_time,
                      const int batch_size,
                      const int beam_width,
                      const int* step_ids,
                      const int* parent_ids,
                      const int end_token,
                      hipStream_t stream)
{
    int batchbeam = batch_size * beam_width;
    dim3 grid(1), block(batchbeam);
    // though decoder do not support > 1024 for now
    if (batchbeam > 1024) {
        grid.x = ceil(batch_size * beam_width / 1024.);
        block.x = 1024;
    }
    gatherTree<<<grid, block, 0, stream>>>(
        beams, max_sequence_lengths, max_time, batch_size, beam_width, step_ids, parent_ids, end_token);
}

__global__ void minusUnfinishedSeqlen(int* sequence_lengths, const bool* finished, const int token_num)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < token_num; i += blockDim.x * gridDim.x) {
        if (finished[i] == false) {
            sequence_lengths[i] -= 1;
        }
    }
}

void invokeMinusUnfinishedSeqlen(int* sequence_lengths, const bool* finished, const int token_num, hipStream_t stream)
{
    dim3 block(min(256, token_num));
    dim3 grid(ceil(token_num / 256.));
    minusUnfinishedSeqlen<<<block, grid, 0, stream>>>(sequence_lengths, finished, token_num);
}

}  // namespace fastertransformer
